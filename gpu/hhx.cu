#include "hip/hip_runtime.h"
/* Created by Language version: 7.1.0 cacheloop */
/* VECTORIZED */
#include <stdio.h>
#include <math.h>
#include "scoplib.h"
#undef PI
#define nil 0
 
#include "md1redef.h"
#include "section.h"
#include "md2redef.h"

#if METHOD3
extern int _method3;
#endif

#include "nrncuda.h"

//#undef exp
//#define exp hoc_Exp
//extern double hoc_Exp(double d);
 
#if !defined(CACHELOOP)
#define CACHELOOP GPU_PODSIZE 
#endif
#if CACHELOOP
#define _cldec int _clj;
#define _cldecjj int _cljj=0, _clss=1;
#define _cldec0 int _clj = 0;
#define _cloff * _cls + _clj
#define _cljarg _clj,
#define _cljproto int _clj,
#define _cls CACHELOOP
#define _clb for (_clj = 0; _clj < _cls; ++_clj) {
#define _cle }
#else
 
#undef CACHELOOP
#define CACHELOOP 0
#define _cldec /**/
#define _cldecjj /**/
#define _cldec0 /**/
#define _cloff /**/
#define _cljarg /**/
#define _cljproto /**/
#define _clj 0
#define _cls 1
#define _cljj 0
#define _clss 1
#define _clb /**/
#define _cle /**/
#endif
 
typedef void(*Pvmi)(_NrnThread* _nt, _Memb_list* _ml, int);


#define _threadargscomma_ _p, _ppvar, _cljarg _thread, _nt,
#define _threadargscommakern_ _dp, _p, _ppvar, _clj,
#define _threadargs_ _p, _ppvar, _cljarg _thread, _nt
#define _threadargskern_ _dp, _p, _ppvar, _clj
 
#define _threadargsprotocomma_ double* _p, Datum* _ppvar, _cljproto Datum* _thread, _NrnThread* _nt,
#define _threadargsprotocommakern_ nrncuda_defines_t _dp, double* _p, Datum* _ppvar, int _clj, 
#define _threadargsproto_ double* _p, Datum* _ppvar, _cljproto Datum* _thread, _NrnThread* _nt
#define _threadargsprotokern_ nrncuda_defines_t _dp, double* _p, Datum* _ppvar, int _clj
 	/*SUPPRESS 761*/
	/*SUPPRESS 762*/
	/*SUPPRESS 763*/
	/*SUPPRESS 765*/
	 extern double *getarg();
 /* Thread safe. No static _p or _ppvar. */
 
#define t _nt->_t
#define dt _nt->_dt
#define gnabar _p[0 _cloff]
#define gkbar _p[1 _cloff]
#define gl _p[2 _cloff]
#define el _p[3 _cloff]
#define gna _p[4 _cloff]
#define gk _p[5 _cloff]
#define il _p[6 _cloff]
#define m _p[7 _cloff]
#define h _p[8 _cloff]
#define n _p[9 _cloff]
#define Dm _p[10 _cloff]
#define Dh _p[11 _cloff]
#define Dn _p[12 _cloff]
#define ena _p[13 _cloff]
#define ek _p[14 _cloff]
#define ina _p[15 _cloff]
#define ik _p[16 _cloff]
#define v _p[17 _cloff]
#define _g _p[18 _cloff]
#define mtau _p[19 _cloff] 
#define minf _p[20 _cloff] 
#define htau _p[21 _cloff] 
#define hinf _p[22 _cloff] 
#define ntau _p[23 _cloff] 
#define ninf _p[24 _cloff] 
#define _ion_ena	_dp._params[_ppvar[0 _cloff]._i]
#define _ion_ina	_dp._params[_ppvar[1 _cloff]._i]
#define _ion_dinadv	_dp._params[_ppvar[2 _cloff]._i]
#define _ion_ek		_dp._params[_ppvar[3 _cloff]._i]
#define _ion_ik		_dp._params[_ppvar[4 _cloff]._i]
#define _ion_dikdv	_dp._params[_ppvar[5 _cloff]._i]
 
#if MAC
#if !defined(v)
#define v _mlhv
#endif
#if !defined(h)
#define h _mlhh
#endif
#endif

extern "C" {
 static int hoc_nrnpointerindex =  -1;
 static Datum* _extcall_thread;
 static Prop* _extcall_prop;
 /* external NEURON variables */
 extern double celsius;
 /* declaration of user functions */
 static int _hoc_rates();
 static int _hoc_vtrap();
 extern int ret(double);
 static int _mechtype;

extern int nrn_get_mechtype(const char*);
extern void _nrn_cacheloop_reg(int type, int cls);
extern void hoc_register_prop_size(int type, int _psize, int _dsize);
extern void hoc_register_cuda_capable(int _type, int _capable);
extern void hoc_register_var(DoubScal* scdoub, DoubVec* vdoub, IntFunc* function);
extern void ivoc_help(char* p);
extern void hoc_register_limits(int type, HocParmLimits* limits);
extern void hoc_register_units( int type, HocParmUnits* units);
extern nrncuda_memb_prop_t nrncuda_memb_prop[30];
extern void nrn_promote(Prop* p, int conc, int rev);

static int _hoc_setdata() {
 Prop *_prop, *hoc_getdata_range(int);
 _prop = hoc_getdata_range(_mechtype);
 _extcall_prop = _prop;
 ret(1.);
 return 1;
}
 /* connect user functions to hoc names */
 static IntFunc hoc_intfunc[] = {
 "setdata_hhx", _hoc_setdata,
 "rates_hhx", _hoc_rates,
 "vtrap_hhx", _hoc_vtrap,
 0, 0
};
#define vtrap vtrap_hhx
static double vtrap ( _threadargsprotocomma_ double _lx , double _ly );

static void _check_rates(_threadargsproto_); 
static void _check_table_thread(double* _p, Datum* _ppvar, Datum* _thread, _NrnThread* _nt, int _type) {
  _cldec0
  // _check_rates(_threadargs_);
 }
 /* declare global and static user variables */
 static int _thread1data_inuse = 0;
static double _thread1data[6*_cls];

#define _gth 0
#define usetable usetable_hhx

 double usetable = 1;
 /* some parameters have upper and lower limits */
 static HocParmLimits _hoc_parm_limits[] = {
 "gl_hhx", 0, 1e+09,
 "gkbar_hhx", 0, 1e+09,
 "gnabar_hhx", 0, 1e+09,
 "usetable_hhx", 0, 1,
 0,0,0
};
 static HocParmUnits _hoc_parm_units[] = {
 "mtau_hhx", "ms",
 "htau_hhx", "ms",
 "ntau_hhx", "ms",
 "gnabar_hhx", "S/cm2",
 "gkbar_hhx", "S/cm2",
 "gl_hhx", "S/cm2",
 "el_hhx", "mV",
 "gna_hhx", "S/cm2",
 "gk_hhx", "S/cm2",
 "il_hhx", "mA/cm2",
 0,0
};
 static double delta_t = 0.01;
 static double h0 = 0;
 static double m0 = 0;
 static double n0 = 0;

static double minf_hhx=0;
static double mtau_hhx=0;
static double hinf_hhx=0;
static double htau_hhx=0;
static double ninf_hhx=0;
static double ntau_hhx=0;

 /* connect global user variables to hoc */
 static DoubScal hoc_scdoub[] = {
 "minf_hhx", &minf_hhx,
 "hinf_hhx", &hinf_hhx,
 "ninf_hhx", &ninf_hhx,
 "mtau_hhx", &mtau_hhx,
 "htau_hhx", &htau_hhx,
 "ntau_hhx", &ntau_hhx,
 "usetable_hhx", &usetable_hhx,
 0,0
};
 static DoubVec hoc_vdoub[] = {
 0,0,0
};
static double _sav_indep;

#define _cvode_ieq _ppvar[6]._i
 /* connect range variables in _p that hoc is supposed to know about */
 static char *_mechanism[] = {
 "7.2.0 nrncuda",
"hhx",
 "gnabar_hhx",
 "gkbar_hhx",
 "gl_hhx",
 "el_hhx",
 0,
 "gna_hhx",
 "gk_hhx",
 "il_hhx",
 0,
 "m_hhx",
 "h_hhx",
 "n_hhx",
 0,
 0};

static void nrn_alloc(Prop* _prop);
static void nrn_cur(_NrnThread* _nt, _Memb_list* _ml, int _type) ;
static void nrn_jacob(_NrnThread* _nt, _Memb_list* _ml, int _type) ;
static void nrn_state(_NrnThread* _nt, _Memb_list* _ml, int _type) ;
static void nrn_init(_NrnThread* _nt, _Memb_list* _ml, int _type) ;

static Pfri _ode_count(int _type);
static Pfri _ode_map(int _ieq, double **_pv, double **_pvdot, double *_pp, Datum* _ppd, double *_atol, int _type); 
static Pfri _ode_matsol(_NrnThread* _nt, _Memb_list* _ml, int _type);
static Pfri _ode_spec(_NrnThread* _nt, _Memb_list* _ml, int _type);

__device__ double atomicAdd(double* address, double val)
{
    double old = *address, assumed;
    do {
        assumed = old;
        old =
           __longlong_as_double(
                  atomicCAS((unsigned long long int*)address,
                            __double_as_longlong(assumed),
                            __double_as_longlong(assumed + val)));
    } while (assumed != old);
    return old;
}

static Symbol* _na_sym;
static Symbol* _k_sym;
 
Prop* need_memb_cl(Symbol* sym, int* cls, int* clj);

static void nrn_alloc(Prop *_prop)
{
	Prop *prop_ion, *need_memb();
	double *_p; Datum *_ppvar;
 	_cldec0 _cldecjj
 
	_p = nrn_prop_data_alloc_cl(_mechtype, 25, _prop, &_clj, &_ppvar, 7);

 	/*initialize range parameters*/
 	gnabar = 0.12;
 	gkbar = 0.036;
 	gl = 0.0003;
 	el = -54.3;
 	_prop->param = _p;
 	_prop->param_size = 25;

        nrncuda_memb_prop[_mechtype].param_size = 25;

 	_prop->dparam = _ppvar;
 	/*connect ionic variables to this model*/
 
	prop_ion = need_memb_cl(_na_sym, &_clss, &_cljj);
 
 nrn_promote(prop_ion, 0, 1);
 	_ppvar[0*_cls + _clj]._pval = &prop_ion->param[0*_clss + _cljj]; /* ena */
 	_ppvar[1*_cls + _clj]._pval = &prop_ion->param[3*_clss + _cljj]; /* ina */
 	_ppvar[2*_cls + _clj]._pval = &prop_ion->param[4*_clss + _cljj]; /* _ion_dinadv */
 
	prop_ion = need_memb_cl(_k_sym, &_clss, &_cljj);

 nrn_promote(prop_ion, 0, 1);
 	_ppvar[3*_cls + _clj]._pval = &prop_ion->param[0*_clss + _cljj]; /* ek */
 	_ppvar[4*_cls + _clj]._pval = &prop_ion->param[3*_clss + _cljj]; /* ik */
 	_ppvar[5*_cls + _clj]._pval = &prop_ion->param[4*_clss + _cljj]; /* _ion_dikdv */
 
	// printf("Finishing hhx alloc\n");
}

 static void _initlists();
  /* some states have an absolute tolerance */
 static Symbol** _atollist;
 static HocStateTolerance _hoc_state_tol[] = {
 0,0
};
 static void _thread_mem_init(Datum*);
 static void _thread_cleanup(Datum*);
 static void _update_ion_pointer(Datum*);
 extern void ion_reg(char *name, double valence);
 extern Symbol* hoc_lookup(char* s);
 extern void register_mech(char**, void(*)(Prop*), Pvmi, Pvmi, Pvmi, Pvmi, int, int);
 extern void _nrn_thread_reg(int i, int cons, void(*f)(Datum*));
 extern void _nrn_thread_table_reg(int i, void(*f)(double* _p, Datum* _ppvar, Datum* _thread, _NrnThread* _nt, int _type));
 extern void hoc_register_cvode(int i, Pfri cnt, Pfri map, Pfri spec, Pfri matsol);
 extern void hoc_register_tolerance(int type, HocStateTolerance* tol, Symbol*** stol);
 extern void _cvode_abstol( Symbol** s, double* tol, int i);

void  _hhx_reg() {
	int _vectorized = 1;
        _initlists();
 	ion_reg("na", -10000.);
 	ion_reg("k", -10000.);
 	_na_sym = hoc_lookup("na_ion");
 	_k_sym = hoc_lookup("k_ion");
 	register_mech(_mechanism, nrn_alloc,nrn_cur, nrn_jacob, nrn_state, nrn_init, hoc_nrnpointerindex, 2);
  _extcall_thread = (Datum*)ecalloc(1, sizeof(Datum));
  _thread_mem_init(_extcall_thread);
  _thread1data_inuse = 0;
 _mechtype = nrn_get_mechtype(_mechanism[1]);
 _nrn_cacheloop_reg(_mechtype, _cls);
     _nrn_thread_reg(_mechtype, 1, _thread_mem_init);
     _nrn_thread_reg(_mechtype, 0, _thread_cleanup);
     _nrn_thread_reg(_mechtype, 2, _update_ion_pointer);
     _nrn_thread_table_reg(_mechtype, _check_table_thread);
  hoc_register_prop_size(_mechtype, 25, 7);
  hoc_register_cuda_capable(_mechtype, 1);
 	hoc_register_cvode(_mechtype,(Pfri) _ode_count,(Pfri) _ode_map,(Pfri) _ode_spec,(Pfri) _ode_matsol);
 	hoc_register_tolerance(_mechtype, _hoc_state_tol, &_atollist);
 	hoc_register_var(hoc_scdoub, hoc_vdoub, hoc_intfunc);
 	ivoc_help("help ?1 hhx ./hhx.mod\n");
 hoc_register_limits(_mechtype, _hoc_parm_limits);
 hoc_register_units(_mechtype, _hoc_parm_units);
 }

 static double *_t_minf;
 static double *_t_mtau;
 static double *_t_hinf;
 static double *_t_htau;
 static double *_t_ninf;
 static double *_t_ntau;
static int _reset;
static char *modelname = "hhx.mod   squid sodium, potassium, and leak channels";

static int error;
static int _ninits = 0;
static int _match_recurse=1;
static void _modl_cleanup(){ _match_recurse=1;}
static int  _f_rates ( _threadargsprotocomma_ double _lv );
static void rates(_threadargsprotocomma_ double _lv); 
 
 static void _n_rates(_threadargsprotocomma_ double _lv);
 static int _slist1[3], _dlist1[3];
 
/*CVODE*/
 static int _ode_spec1 (_threadargsproto_) {int _reset = 0; {
   _clb rates ( _threadargscomma_ v ) ;
   _cle _clb Dm = ( minf - m ) / mtau ;
   _cle _clb Dh = ( hinf - h ) / htau ;
   _cle _clb Dn = ( ninf - n ) / ntau ;
   _cle }
 return _reset;
}
 static int _ode_matsol1 (_threadargsproto_) {
 _clb rates ( _threadargscomma_ v ) ;
 _cle _clb Dm = Dm  / (1. - dt*( ( ( ( - 1.0 ) ) ) / mtau )) ;
 _cle _clb Dh = Dh  / (1. - dt*( ( ( ( - 1.0 ) ) ) / htau )) ;
 _cle _clb Dn = Dn  / (1. - dt*( ( ( ( - 1.0 ) ) ) / ntau )) ; _cle
return 0;
}
 /*END CVODE*/

__device__ void rates_k(_threadargsprotocommakern_ double _lv);
__device__ void states_k (_threadargsprotokern_, double dt_k) { 
   rates_k ( _threadargscommakern_ v ) ;
   m = m + (1. - exp(dt_k*(( ( ( - 1.0 ) ) ) / mtau)))*(- ( ( ( minf ) ) / mtau ) / ( ( ( ( - 1.0) ) ) / mtau ) - m) ;
   h = h + (1. - exp(dt_k*(( ( ( - 1.0 ) ) ) / htau)))*(- ( ( ( hinf ) ) / htau ) / ( ( ( ( - 1.0) ) ) / htau ) - h) ;
   n = n + (1. - exp(dt_k*(( ( ( - 1.0 ) ) ) / ntau)))*(- ( ( ( ninf ) ) / ntau ) / ( ( ( ( - 1.0) ) ) / ntau ) - n) ;
}


static int states (_threadargsproto_) { {
   _clb rates ( _threadargscomma_ v ) ;
   _cle _clb  m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mtau)))*(- ( ( ( minf ) ) / mtau ) / ( ( ( ( - 1.0) ) ) / mtau ) - m) ;
   _cle _clb  h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / htau)))*(- ( ( ( hinf ) ) / htau ) / ( ( ( ( - 1.0) ) ) / htau ) - h) ;
   _cle _clb  n = n + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / ntau)))*(- ( ( ( ninf ) ) / ntau ) / ( ( ( ( - 1.0) ) ) / ntau ) - n) ;
   _cle }
  return 0;
}

static double _mfac_rates, _tmin_rates;


static void _check_rates(_threadargsproto_) {
  static int _maktable=1; int _i, _j, _ix = 0;
  double _xi, _tmax;
  static double _sav_celsius;
  if (!usetable) {return;}
  if (_sav_celsius != celsius) { _maktable = 1;}
  if (_maktable) { 
   double _x, _dx;
    _maktable=0;
   _tmin_rates =  - 100.0 ;
   _tmax =  100.0 ;
   _dx = (_tmax - _tmin_rates)/200.;
    _mfac_rates = 1./_dx;
   for (_i=0, _x=_tmin_rates; _i < 201; _x += _dx, _i++) {
    _f_rates(_threadargscomma_ _x);
    _t_minf[_i] = minf;
    _t_mtau[_i] = mtau;
    _t_hinf[_i] = hinf;
    _t_htau[_i] = htau;
    _t_ninf[_i] = ninf;
    _t_ntau[_i] = ntau;
   }
   _sav_celsius = celsius;
  }
 }

static void rates(_threadargsprotocomma_ double _lv) { 
#if 0
_check_rates(_threadargs_);
#endif
 _n_rates(_threadargscomma_ _lv);
 return;
 }

__device__ void _f_rates_k(_threadargsprotocommakern_ double _lv);
__device__ void rates_k(_threadargsprotocommakern_ double _lv) { 
//#if 0
//_check_rates(_threadargs_);
//#endif
// _n_rates(_threadargscomma_ _lv);
_f_rates_k(_threadargscommakern_ _lv);
}

 static void _n_rates(_threadargsprotocomma_ double _lv){
 int _i, _j;
 double _xi, _theta;
 if (!usetable) {
    _f_rates(_threadargscomma_ _lv); return; 
 }
 _xi = _mfac_rates * (_lv - _tmin_rates);
 _i = (int) _xi;
 if (_xi <= 0.) {
 minf = _t_minf[0];
 mtau = _t_mtau[0];
 hinf = _t_hinf[0];
 htau = _t_htau[0];
 ninf = _t_ninf[0];
 ntau = _t_ntau[0];
 return; }
 if (_xi >= 200.) {
 minf = _t_minf[200];
 mtau = _t_mtau[200];
 hinf = _t_hinf[200];
 htau = _t_htau[200];
 ninf = _t_ninf[200];
 ntau = _t_ntau[200];
 return; }
 _theta = _xi - (double)_i;
 minf = _t_minf[_i] + _theta*(_t_minf[_i+1] - _t_minf[_i]);
 mtau = _t_mtau[_i] + _theta*(_t_mtau[_i+1] - _t_mtau[_i]);
 hinf = _t_hinf[_i] + _theta*(_t_hinf[_i+1] - _t_hinf[_i]);
 htau = _t_htau[_i] + _theta*(_t_htau[_i+1] - _t_htau[_i]);
 ninf = _t_ninf[_i] + _theta*(_t_ninf[_i+1] - _t_ninf[_i]);
 ntau = _t_ntau[_i] + _theta*(_t_ntau[_i+1] - _t_ntau[_i]);
 }

__device__ double vtrap_k (_threadargsprotocommakern_ double _lx , double _ly);
__device__ void  _f_rates_k (_threadargsprotocommakern_ double _lv)
 {
   double _lalpha , _lbeta , _lsum , _lq10 ;
  _lq10 = pow( 3.0 , ( ( _dp.celsius - 6.3 ) / 10.0 ) ) ;
   _lalpha = .1 * vtrap_k ( _threadargscommakern_ - ( _lv + 40.0 ) , 10.0 ) ;
   _lbeta = 4.0 * exp ( - ( _lv + 65.0 ) / 18.0 ) ;
   _lsum = _lalpha + _lbeta ;
   mtau = 1.0 / ( _lq10 * _lsum ) ;
   minf = _lalpha / _lsum ;
   _lalpha = .07 * exp ( - ( _lv + 65.0 ) / 20.0 ) ;
   _lbeta = 1.0 / ( exp ( - ( _lv + 35.0 ) / 10.0 ) + 1.0 ) ;
   _lsum = _lalpha + _lbeta ;
   htau = 1.0 / ( _lq10 * _lsum ) ;
   hinf = _lalpha / _lsum ;
   _lalpha = .01 * vtrap_k ( _threadargscommakern_ - ( _lv + 55.0 ) , 10.0 ) ;
   _lbeta = .125 * exp ( - ( _lv + 65.0 ) / 80.0 ) ;
   _lsum = _lalpha + _lbeta ;
   ntau = 1.0 / ( _lq10 * _lsum ) ;
   ninf = _lalpha / _lsum ;
}


static int  _f_rates ( _threadargsprotocomma_ double _lv ) 
  //  double* _p; Datum* _ppvar; _cldec Datum* _thread; _NrnThread* _nt; 
  //  double _lv ;
 {
   double _lalpha , _lbeta , _lsum , _lq10 ;
  _lq10 = pow( 3.0 , ( ( celsius - 6.3 ) / 10.0 ) ) ;
   _lalpha = .1 * vtrap ( _threadargscomma_ - ( _lv + 40.0 ) , 10.0 ) ;
   _lbeta = 4.0 * exp ( - ( _lv + 65.0 ) / 18.0 ) ;
   _lsum = _lalpha + _lbeta ;
   mtau = 1.0 / ( _lq10 * _lsum ) ;
   minf = _lalpha / _lsum ;
   _lalpha = .07 * exp ( - ( _lv + 65.0 ) / 20.0 ) ;
   _lbeta = 1.0 / ( exp ( - ( _lv + 35.0 ) / 10.0 ) + 1.0 ) ;
   _lsum = _lalpha + _lbeta ;
   htau = 1.0 / ( _lq10 * _lsum ) ;
   hinf = _lalpha / _lsum ;
   _lalpha = .01 * vtrap ( _threadargscomma_ - ( _lv + 55.0 ) , 10.0 ) ;
   _lbeta = .125 * exp ( - ( _lv + 65.0 ) / 80.0 ) ;
   _lsum = _lalpha + _lbeta ;
   ntau = 1.0 / ( _lq10 * _lsum ) ;
   ninf = _lalpha / _lsum ;
    return 0; 
}
 
static int _hoc_rates() {
  double _r;
   double* _p; Datum* _ppvar; _cldec Datum* _thread; _NrnThread* _nt;
   if (_extcall_prop) {
	_p = _extcall_prop->param; 
	_ppvar = _extcall_prop->dparam;
   }else{
        _p = (double*)0;
	 _ppvar = (Datum*)0;
  }
  _thread = _extcall_thread;
  _nt = nrn_threads;
 
#if 1
 _check_rates(_threadargs_);
#endif
 _r = 1.;
 rates ( _threadargscomma_ *getarg(1) ) ;
 ret(_r);
return 0;
}
 
__device__ double vtrap_k ( _threadargsprotocommakern_ double _lx , double _ly ) 
{
   double _lvtrap;
   if ( fabs ( _lx / _ly ) < 1e-6 ) {
     _lvtrap = _ly * ( 1.0 - _lx / _ly / 2.0 ) ;
     }
   else {
     _lvtrap = _lx / ( exp ( _lx / _ly ) - 1.0 ) ;
     }
   return _lvtrap;
}
 
double vtrap ( _threadargsprotocomma_ double _lx , double _ly )
 {
   double _lvtrap;
 if ( fabs ( _lx / _ly ) < 1e-6 ) {
     _lvtrap = _ly * ( 1.0 - _lx / _ly / 2.0 ) ;
     }
   else {
     _lvtrap = _lx / ( exp ( _lx / _ly ) - 1.0 ) ;
     }
   
return _lvtrap;
 }
 
static int _hoc_vtrap() {
  double _r;
   double* _p; Datum* _ppvar; _cldec Datum* _thread; _NrnThread* _nt;
   if (_extcall_prop) {_p = _extcall_prop->param; _ppvar = _extcall_prop->dparam;}else{ _p = (double*)0; _ppvar = (Datum*)0; }
  _thread = _extcall_thread;
  _nt = nrn_threads;
 _r =  vtrap ( _threadargscomma_ *getarg(1) , *getarg(2) ) ;
 ret(_r);
 return 0;
}
 
static Pfri _ode_count(int _type) { return (Pfri)3;}
 
static Pfri _ode_spec(_NrnThread* _nt, _Memb_list* _ml, int _type) {
   double* _p; Datum* _ppvar; Datum* _thread;
   Node* _nd; int _iml, _cntml; _cldec
  _cntml = _ml->_nodecount;
  _thread = _ml->_thread;
  for (_iml = 0; _iml < _cntml; ++_iml) {
    _p = _ml->_data[_iml]; _ppvar = _ml->_pdata[_iml];
    _clb _nd = _ml->_nodelist[_iml*_cls + _clj];
    v = NODEV(_nd); _cle
/* cmc
  _clb ena = _ion_ena; _cle
  _clb ek = _ion_ek; _cle
*/
     _ode_spec1 (_threadargs_);
   }
return 0;
}
 
static Pfri _ode_map(int _ieq, double **_pv, double **_pvdot, double *_pp, Datum* _ppd, double *_atol, int _type) 
{ 
	double* _p; Datum* _ppvar; _cldec0
 	int _i; _p = _pp; _ppvar = _ppd;
	_cvode_ieq = _ieq;
	for (_i=0; _i < 3; ++_i) {
		_pv[_i] = _pp + _slist1[_i];  _pvdot[_i] = _pp + _dlist1[_i];
		_cvode_abstol(_atollist, _atol, _i);
	}
	return 0;
 }
 
static Pfri _ode_matsol(_NrnThread* _nt, _Memb_list* _ml, int _type) {
   double* _p; Datum* _ppvar; Datum* _thread;
   Node* _nd; int _iml, _cntml; _cldec
  _cntml = _ml->_nodecount;
  _thread = _ml->_thread;
  for (_iml = 0; _iml < _cntml; ++_iml) {
    _p = _ml->_data[_iml]; _ppvar = _ml->_pdata[_iml];
    _clb _nd = _ml->_nodelist[_iml*_cls + _clj];
    v = NODEV(_nd); _cle
  /* cmc _clb ena = _ion_ena; _cle
  _clb ek = _ion_ek; _cle
*/
 _ode_matsol1 (_threadargs_);
 }
return 0;
}
 
static void _thread_mem_init(Datum* _thread) {
  if (_thread1data_inuse) {
     _thread[_gth]._pval = (double*)ecalloc(6*_cls, sizeof(double));
 }else{
      _thread[_gth]._pval = _thread1data; _thread1data_inuse = 1;
 }
 }
 
static void _thread_cleanup(Datum* _thread) {
  if (_thread[_gth]._pval == _thread1data) {
   _thread1data_inuse = 0;
  }else{
   free((void*)_thread[_gth]._pval);
  }
 }

 extern void nrn_update_ion_pointer(Symbol*, Datum*, int, int, int);
 static void _update_ion_pointer(Datum* _ppvar) {

   nrn_update_ion_pointer(_na_sym, _ppvar, 0, 0, _cls);
   nrn_update_ion_pointer(_na_sym, _ppvar, 1, 3, _cls);
   nrn_update_ion_pointer(_na_sym, _ppvar, 2, 4, _cls);
   nrn_update_ion_pointer(_k_sym, _ppvar, 3, 0, _cls);
   nrn_update_ion_pointer(_k_sym, _ppvar, 4, 3, _cls);
   nrn_update_ion_pointer(_k_sym, _ppvar, 5, 4, _cls);
 }


__device__ void _hhx_initmodel_k(_threadargsprotokern_)
{
  // h = h0;
  // m = m0; 
  // n = n0; 

  rates_k ( _threadargscommakern_ v);
  m = minf ;
  h = hinf ;
  n = ninf ;
}

/*
static void initmodel(_threadargsproto_) {
  int _i; double _save;{
  _clb h = h0; _cle
  _clb m = m0; _cle
  _clb n = n0; _cle
 {
   _clb rates ( _threadargscomma_ v ) ;
   _cle _clb m = minf ;
   _cle _clb h = hinf ;
   _cle _clb n = ninf ;
   _cle }
 
}
}
*/

  
__global__ void
gpu_hhx_init_kernel(nrncuda_defines_t _dp, nrncuda_memb_prop_t mp, double t_k, double dt_k)
{
        double _rhs;
        _cldec

        int _pod  = (blockIdx.x*GPU_ADVANCE_BLOCKSIZE/GPU_PODSIZE) + (threadIdx.x >> GPU_LOGPODSIZE);
	if (_pod >= mp.num_pods) return;
        _clj  =  threadIdx.x & (GPU_PODSIZE-1);
        int podoffset = GPU_PODSIZE * _pod;
        double* _p = &_dp._params[mp.param_start_offset+podoffset*25];
        Datum* _ppvar = &_dp._dparams[mp.dparam_start_offset+podoffset*7];
        int node = _dp._nodeindices[mp.node_start_offset+podoffset + _clj];

 	v = _dp.VEC._v[node];
        ena = _ion_ena; 
        ek  = _ion_ek; 
        _hhx_initmodel_k(_threadargskern_);
        return;
}

static void nrn_init(_NrnThread* _nt, _Memb_list* _ml, int _type){
        hipError_t err;
        int num_blocks;
        int num;

        /* need to work out an optimal strategy for setting up the grid based on the number of
         compartments and the number of mechanisms. But for starters, we will just use
         number of nodes. */

	nrncuda_memb_prop[_mechtype].num_pods = _ml->_nodecount;
        num_blocks = _ml->_nodecount * GPU_PODSIZE / GPU_ADVANCE_BLOCKSIZE;
        num = num_blocks * GPU_ADVANCE_BLOCKSIZE;
        if (_ml->_nodecount * GPU_PODSIZE > num) num_blocks += 1;
	nrncuda_memb_prop[_mechtype].num_blocks = num_blocks;

        dim3 dimBlock(GPU_ADVANCE_BLOCKSIZE, 1, 1);
        dim3 dimGrid(nrncuda_memb_prop[_mechtype].num_blocks, 1, 1);

	_nt->nrncuda_defines.celsius = celsius;
        gpu_hhx_init_kernel<<<dimGrid, dimBlock>>>(_nt->nrncuda_defines, nrncuda_memb_prop[_mechtype], _nt->_t, _nt->_dt);
        err = hipGetLastError();
 
        // printf("Done with pas init\n");
}

/*
static void nrn_init(_NrnThread* _nt, _Memb_list* _ml, int _type){
double* _p; Datum* _ppvar; Datum* _thread; _cldec
Node *_nd; int* _ni; int _iml, _cntml;
   _ni = _ml->_nodeindices;
   _cntml = _ml->_nodecount;
   _thread = _ml->_thread;
   for (_iml = 0; _iml < _cntml; ++_iml) {
      _p = _ml->_data[_iml]; _ppvar = _ml->_pdata[_iml];

#if 0
 _check_rates(_threadargs_);
#endif
     _clb v = VEC_V(_ni[_iml*_cls + _clj]); _cle
     _clb ena = _ion_ena; _cle
     _clb ek = _ion_ek; _cle
     initmodel(_threadargs_);
  }}
*/


static void _nrn_current(_threadargsprotocomma_ double* _current){ _clb _current[_clj] = 0.; _cle
{ {
   _clb gna = gnabar * m * m * m * h ;
   _cle _clb ina = gna * ( v - ena ) ;
   _cle _clb gk = gkbar * n * n * n * n ;
   _cle _clb ik = gk * ( v - ek ) ;
   _cle _clb il = gl * ( v - el ) ;
   _cle }
 _clb _current[_clj] += ina; _cle
 _clb _current[_clj] += ik; _cle
 _clb _current[_clj] += il; _cle

}}

/*
__device__ void _hhx_current(_threadargsprotocommakern_ double* _current){
   *_current = 0.;
   gna = gnabar * m * m * m * h ;
   ina = gna * ( v - ena ) ;
   gk = gkbar * n * n * n * n ;
   ik = gk * ( v - ek ) ;
   il = gl * ( v - el ) ;
   *_current += ina; 
   *_current += ik; 
   *_current += il;
}
*/

#define _hhx_current(_curr) \
   _curr = 0.; \
   _lgna = gnabar * m * m * m * h ; \
   _lina = _lgna * ( _lv - _lena ) ; \
   _lgk =  gkbar * n * n * n * n ; \
   _lik = _lgk * ( _lv - _lek ) ; \
   _lil = _lgl * ( _lv - _lel ) ; \
   _curr += _lina; \
   _curr += _lik; \
   _curr += _lil; \

__global__ void
gpu_hhx_cur_kernel(nrncuda_defines_t _dp, nrncuda_memb_prop_t mp)
{
        double _lrhs;
        _cldec

	double _lv;
	double _lgna;
	double _lgk;
	double _lgl;
	double _lena;
	double _lek;
	double _lel;

	double _lik;
	double _lina;
	double _lil;

        int _pod  = (blockIdx.x*GPU_ADVANCE_BLOCKSIZE/GPU_PODSIZE) + (threadIdx.x >> GPU_LOGPODSIZE);
	if (_pod >= mp.num_pods) return;
        _clj  =  threadIdx.x & (GPU_PODSIZE-1);
        int podoffset = GPU_PODSIZE * _pod;
        double* _p = &_dp._params[mp.param_start_offset+podoffset*25];
        Datum* _ppvar = &_dp._dparams[mp.dparam_start_offset+podoffset*7];
        int node = _dp._nodeindices[mp.node_start_offset+podoffset + _clj];

        _lv = _dp.VEC._v[node];
  
  	_lena = _ion_ena;
	_lek = _ion_ek; 
	_lel = el; 
	_lgl = gl; 
	_lv += .001; 
	_hhx_current(_g);
 	_lv -= .001; 
 	double _dik = _lik;
 	double _dina = _lina;
	_hhx_current(_lrhs);
        atomicAdd(&_ion_dinadv, (_dina - _lina)/.001);
        atomicAdd(&_ion_dikdv,  (_dik - _lik)/.001); 
	_g = (_g - _lrhs)/.001; 
        atomicAdd(&_ion_ina,  _lina); 
        atomicAdd(&_ion_ik,   _lik); 
        atomicAdd(&_dp.VEC._rhs[node], -_lrhs);
}

static void nrn_cur(_NrnThread* _nt, _Memb_list* _ml, int _type){
        hipError_t err;

        dim3 dimBlock(GPU_ADVANCE_BLOCKSIZE, 1, 1);
        dim3 dimGrid(nrncuda_memb_prop[_mechtype].num_blocks, 1, 1);

        err = hipMemcpy( _nt->nrncuda_defines.VEC._rhs,  _nt->_actual_rhs,
                          _nt->nrncuda_defines.VEC._size_rhs, hipMemcpyHostToDevice);
        if (err != hipSuccess) return;
        gpu_hhx_cur_kernel<<<dimGrid, dimBlock>>>(_nt->nrncuda_defines, nrncuda_memb_prop[_mechtype]);
        err = hipGetLastError();
        if (err != hipSuccess) return;
        err = hipMemcpy( _nt->_actual_rhs, _nt->nrncuda_defines.VEC._rhs,
                          _nt->nrncuda_defines.VEC._size_rhs, hipMemcpyDeviceToHost);
        if (err != hipSuccess) return;

        // printf("Done with pas cur\n");

}


/*
static void nrn_jacob(_NrnThread* _nt, _Memb_list* _ml, int _type) {
double* _p; Datum* _ppvar; Datum* _thread; _cldec0
Node *_nd; int* _ni; int _iml, _cntml;
#if CACHEVEC
    _ni = _ml->_nodeindices;
#endif
_cntml = _ml->_nodecount;
_thread = _ml->_thread;
for (_iml = 0; _iml < _cntml; ++_iml) {
 _p = _ml->_data[_iml];
#if CACHEVEC
  if (use_cachevec) {
	_clb VEC_D(_ni[_iml*_cls + _clj]) += _g; _cle
  }else
#endif
  {
     _clb _nd = _ml->_nodelist[_iml*_cls + _clj];
	NODED(_nd) += _g; _cle
  }
 
}}
*/

__global__ void
gpu_hhx_jacob_kernel(nrncuda_defines_t _dp, nrncuda_memb_prop_t mp)
{
        double _rhs;
        _cldec

        int _pod  = (blockIdx.x*GPU_ADVANCE_BLOCKSIZE/GPU_PODSIZE) + (threadIdx.x >> GPU_LOGPODSIZE);
	if (_pod >= mp.num_pods) return;
        _clj  =  threadIdx.x & (GPU_PODSIZE-1);
        int podoffset = GPU_PODSIZE * _pod;
        double* _p = &_dp._params[mp.param_start_offset+podoffset*25];
        Datum* _ppvar = &_dp._dparams[mp.dparam_start_offset+podoffset*7];
        int node = _dp._nodeindices[mp.node_start_offset+podoffset + _clj];
        atomicAdd(&_dp.VEC._d[node], _g);
}

static void nrn_jacob(_NrnThread* _nt, _Memb_list* _ml, int _type){
        hipError_t err;

        dim3 dimBlock(GPU_ADVANCE_BLOCKSIZE, 1, 1);
        dim3 dimGrid(nrncuda_memb_prop[_mechtype].num_blocks, 1, 1);

        err = hipMemcpy( _nt->nrncuda_defines.VEC._d,  _nt->_actual_d,
                          _nt->nrncuda_defines.VEC._size_d, hipMemcpyHostToDevice);
        if (err != hipSuccess) return;
        gpu_hhx_jacob_kernel<<<dimGrid, dimBlock>>>(_nt->nrncuda_defines, nrncuda_memb_prop[_mechtype]);
        err = hipGetLastError();
        if (err != hipSuccess) return;
        err = hipMemcpy( _nt->_actual_d, _nt->nrncuda_defines.VEC._d,
                          _nt->nrncuda_defines.VEC._size_d, hipMemcpyDeviceToHost);
        if (err != hipSuccess) return;

        // printf("Done with pas jacob\n");

}

/*
static void nrn_state(_NrnThread* _nt, _Memb_list* _ml, int _type) {
 double _break, _save;
double* _p; Datum* _ppvar; Datum* _thread; _cldec0
Node *_nd; int* _ni; int _iml, _cntml;
#if CACHEVEC
    _ni = _ml->_nodeindices;
#endif
_cntml = _ml->_nodecount;
_thread = _ml->_thread;
for (_iml = 0; _iml < _cntml; ++_iml) {
 _p = _ml->_data[_iml]; _ppvar = _ml->_pdata[_iml];
 _nd = _ml->_nodelist[_iml];
#if CACHEVEC
  if (use_cachevec) {
    _clb v = VEC_V(_ni[_iml*_cls + _clj]); _cle
  }else
#endif
  {
    _clb _nd = _ml->_nodelist[_iml*_cls + _clj]; v = NODEV(_nd); _cle
  }
 _break = t + .5*dt; _save = t;

{
  _clb ena = _ion_ena; _cle
  _clb ek = _ion_ek; _cle
 { {
 for (; t < _break; t += dt) {_cldec0
   states(_threadargs_);
}}
 t = _save;
 }  }}

}
*/

__global__ void
gpu_hhx_state_kernel(nrncuda_defines_t _dp, nrncuda_memb_prop_t mp, double tk, double dtk)
{
        double _rhs;
        _cldec
	double _break;
	double _save;
	double _lt;
	double _lv;

        int _pod  = (blockIdx.x*GPU_ADVANCE_BLOCKSIZE/GPU_PODSIZE) + (threadIdx.x >> GPU_LOGPODSIZE);
	if (_pod >= mp.num_pods) return;
        _clj  =  threadIdx.x & (GPU_PODSIZE-1);
        int podoffset = GPU_PODSIZE * _pod;
        double* _p = &_dp._params[mp.param_start_offset+podoffset*25];
        Datum* _ppvar = &_dp._dparams[mp.dparam_start_offset+podoffset*7];
        int node = _dp._nodeindices[mp.node_start_offset+podoffset + _clj];

        v = _dp.VEC._v[node];
	_lt = tk;
	_break = tk + .5 * dtk;  _save = tk;
 
	for (; _lt < _break; _lt += dtk) {
	    states_k(_threadargskern_, dtk); 
	}
	/* should t be in dp? */
}

static void nrn_state(_NrnThread* _nt, _Memb_list* _ml, int _type){
        hipError_t err;

        dim3 dimBlock(GPU_ADVANCE_BLOCKSIZE, 1, 1);
        dim3 dimGrid(nrncuda_memb_prop[_mechtype].num_blocks, 1, 1);

        gpu_hhx_state_kernel<<<dimGrid, dimBlock>>>(_nt->nrncuda_defines, nrncuda_memb_prop[_mechtype], _nt->_t,_nt->_dt);
        err = hipGetLastError();

        // printf("Done with pas states\n");

}


static void terminal(){}

static void _initlists(){
 double _x; double* _p = &_x; _cldec0
 int _i; static int _first = 1;
  if (!_first) return;
 _slist1[0] = &(m) - _p;  _dlist1[0] = &(Dm) - _p;
 _slist1[1] = &(h) - _p;  _dlist1[1] = &(Dh) - _p;
 _slist1[2] = &(n) - _p;  _dlist1[2] = &(Dn) - _p;
/*
   _t_minf = makevector(201*sizeof(double));
   _t_mtau = makevector(201*sizeof(double));
   _t_hinf = makevector(201*sizeof(double));
   _t_htau = makevector(201*sizeof(double));
   _t_ninf = makevector(201*sizeof(double));
   _t_ntau = makevector(201*sizeof(double));
*/
_first = 0;
}
}
