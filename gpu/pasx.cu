#include "hip/hip_runtime.h"
/* Created by Language version: 7.1.0 cacheloop */
/* VECTORIZED */
#include <stdio.h>
#include <math.h>
#include "scoplib.h"
#undef PI
#define nil 0
 
#include "md1redef.h"
#include "section.h"
#include "md2redef.h"

#if METHOD3
extern int _method3;
#endif

#include "nrncuda.h"

#undef exp
#define exp hoc_Exp
extern double hoc_Exp();
 
#if !defined(CACHELOOP)
#define CACHELOOP GPU_PODSIZE 
#endif
#if CACHELOOP
#define _cldec int _clj;
#define _cldecjj int _cljj=0, _clss=1;
#define _cldec0 int _clj = 0;
#define _cls CACHELOOP
#define _cloff * _cls + _clj
#define _cljarg _clj,
#define _cljproto int _clj,
#define _clb for (_clj = 0; _clj < _cls; ++_clj) {
#define _cle }
#else
 
#undef CACHELOOP
#define CACHELOOP 0
#define _cldec /**/
#define _cldecjj /**/
#define _cldec0 /**/
#define _cloff /**/
#define _cljarg /**/
#define _cljproto /**/
#define _clj 0
#define _cls 1
#define _cljj 0
#define _clss 1
#define _clb /**/
#define _cle /**/
#endif
 

typedef void(*Pvmi)(_NrnThread* _nt, _Memb_list* _ml, int); 

#define _threadargscomma_ _p, _ppvar, _cljarg _thread, _nt,
#define _threadargs_ _p, _ppvar, _cljarg _thread, _nt
 
#define _threadargsprotocomma_ double* _p, Datum* _ppvar, _cljproto Datum* _thread, _NrnThread* _nt,
#define _threadargsproto_ double* _p, Datum* _ppvar, _cljproto Datum* _thread, _NrnThread* _nt
 	/*SUPPRESS 761*/
	/*SUPPRESS 762*/
	/*SUPPRESS 763*/
	/*SUPPRESS 765*/
	 extern double *getarg();
 /* Thread safe. No static _p or _ppvar. */
 
#define t _nt->_t
#define dt _nt->_dt
#define g _p[0 _cloff]
#define e _p[1 _cloff]
#define i _p[2 _cloff]
#define v _p[3 _cloff]
#define _g _p[4 _cloff]
 
#if MAC
#if !defined(v)
#define v _mlhv
#endif
#if !defined(h)
#define h _mlhh
#endif
#endif

extern "C" {
 static int hoc_nrnpointerindex =  -1;
 static Datum* _extcall_thread;
 static Prop* _extcall_prop;
 /* external NEURON variables */
 /* declaration of user functions */
 extern int ret(double);
 static int _mechtype;

extern int nrn_get_mechtype(const char*);
extern void _nrn_cacheloop_reg(int type, int cls);
extern void hoc_register_prop_size(int type, int _psize, int _dsize);
extern void hoc_register_cuda_capable(int _type, int _capable);
extern void hoc_register_var(DoubScal* scdoub, DoubVec* vdoub, IntFunc* function);
extern void ivoc_help(char* p);
extern void hoc_register_limits(int type, HocParmLimits* limits);
extern void hoc_register_units( int type, HocParmUnits* units);
extern Memb_func* memb_func;


static int _hoc_setdata() {
 Prop *_prop, *hoc_getdata_range(int);
 _prop = hoc_getdata_range(_mechtype);
 _extcall_prop = _prop;
 ret(1.);
 return 1;
}
 /* connect user functions to hoc names */
 static IntFunc hoc_intfunc[] = {
 "setdata_pasx", _hoc_setdata,
 0, 0
};
 /* declare global and static user variables */
 /* some parameters have upper and lower limits */
 static HocParmLimits _hoc_parm_limits[] = {
 "g_pasx", 0, 1e+09,
 0,0,0
};
 static HocParmUnits _hoc_parm_units[] = {
 "g_pasx", "S/cm2",
 "e_pasx", "mV",
 "i_pasx", "mA/cm2",
 0,0
};
 /* connect global user variables to hoc */
 static DoubScal hoc_scdoub[] = {
 0,0
};
 static DoubVec hoc_vdoub[] = {
 0,0,0
};
 static double _sav_indep;
 static void nrn_alloc(Prop* _prop);
 static void nrn_cur(_NrnThread* _nt, _Memb_list* _ml, int _type) ;
 static void nrn_jacob(_NrnThread* _nt, _Memb_list* _ml, int _type) ;
 static void nrn_state(_NrnThread* _nt, _Memb_list* _ml, int _type) ;
 static void nrn_init(_NrnThread* _nt, _Memb_list* _ml, int _type) ;
 /* connect range variables in _p that hoc is supposed to know about */
 static char *_mechanism[] = {
 "7.2.0 nrncuda",
"pasx",
 "g_pasx",
 "e_pasx",
 0,
 "i_pasx",
 0,
 0,
 0};

#define atomicAdd _pasx_atomicAdd
__device__ double _pasx_atomicAdd(double* address, double val)
{
    double old = *address, assumed;
    do {
        assumed = old;
        old =
           __longlong_as_double(
                  atomicCAS((unsigned long long int*)address,
                            __double_as_longlong(assumed),
                            __double_as_longlong(assumed + val)));
    } while (assumed != old);
    return old;
}

static void nrn_alloc(Prop* _prop)
{
	Prop *prop_ion, *need_memb(), *need_memb_cl();
	double *_p; Datum *_ppvar;
 	_cldec0 _cldecjj
 
#if CACHELOOP
	_p = nrn_prop_data_alloc_cl(_mechtype, 5, _prop, &_clj, &_ppvar, 0);
#else
 	_p = nrn_prop_data_alloc(_mechtype, 5, _prop);
 
#endif
 	/*initialize range parameters*/
 	g = 0.001;
 	e = -70;
 	_prop->param = _p;
 	_prop->param_size = 5;

	//printf("Done with nrn_alloc in pasx.cu\n");

}

extern void register_mech(char**, void(*)(Prop*), Pvmi, Pvmi, Pvmi, Pvmi, int, int);
static void _initlists();
void _pasx_reg() {
	int _vectorized = 1;
  _initlists();
 	register_mech(_mechanism, nrn_alloc, nrn_cur, nrn_jacob, nrn_state, nrn_init, hoc_nrnpointerindex, 1);
 _mechtype = nrn_get_mechtype(_mechanism[1]);
 _nrn_cacheloop_reg(_mechtype, _cls);
  hoc_register_prop_size(_mechtype, 5, 0);
  hoc_register_cuda_capable(_mechtype, 1);
 	hoc_register_var(hoc_scdoub, hoc_vdoub, hoc_intfunc);
 	ivoc_help("help ?1 pasx ./pasx.mod\n");
 hoc_register_limits(_mechtype, _hoc_parm_limits);
 hoc_register_units(_mechtype, _hoc_parm_units);
 }

static int _reset;
static char *modelname = "passive membrane channel";

static int error;
static int _ninits = 0;
static int _match_recurse=1;
static void _modl_cleanup(){ _match_recurse=1;}

static void initmodel(_threadargsproto_) {
  int _i; double _save;{
}
}

/*
static void nrn_init(_NrnThread* _nt, _Memb_list* _ml, int _type){
double* _p; Datum* _ppvar; Datum* _thread; _cldec
Node *_nd; int* _ni; int _iml, _cntml;
    _ni = _ml->_nodeindices;
    _cntml = _ml->_nodecount;
    _thread = _ml->_thread;
    for (_iml = 0; _iml < _cntml; ++_iml) {
        _p = _ml->_data[_iml]; _ppvar = _ml->_pdata[_iml];
        _clb v = VEC_V(_ni[_iml*_cls + _clj]); _cle
        initmodel(_threadargs_);
    }
}
*/

/*
__global__ void 
_initmodel(_threadargsproto_) {
  int _i; double _save;{
}
}
*/

__global__ void
gpu_init_kernel(nrncuda_defines_t dp, nrncuda_memb_prop_t mp)
{
	double _rhs;
	Datum* _ppvar;
        _cldec

        int _pod  = (blockIdx.x*GPU_ADVANCE_BLOCKSIZE/GPU_PODSIZE) + (threadIdx.x >> GPU_LOGPODSIZE);
	if (_pod >= mp.num_pods) return;
        _clj  =  threadIdx.x & (GPU_PODSIZE-1);
        int podoffset = GPU_PODSIZE * _pod;
        double* _p = &dp._params[mp.param_start_offset + podoffset*5];
        int node = dp._nodeindices[mp.node_start_offset + podoffset + _clj];

        v = dp.VEC._v[node]; 
}

static void nrn_init(_NrnThread* _nt, _Memb_list* _ml, int _type){
        hipError_t err;
        int num_blocks;
        int num;

        /* need to work out an optimal strategy for setting up the grid based on the number of
         compartments and the number of mechanisms. But for starters, we will just use
         number of nodes. */

	_ml->nrncuda_info->num_pods = _ml->_nodecount;
        num_blocks = _ml->_nodecount * GPU_PODSIZE / GPU_ADVANCE_BLOCKSIZE;
        num = num_blocks * GPU_ADVANCE_BLOCKSIZE;
        if (_ml->_nodecount * GPU_PODSIZE > num) num_blocks += 1;
	_ml->nrncuda_info->num_blocks = num_blocks;

        dim3 dimBlock(GPU_ADVANCE_BLOCKSIZE, 1, 1);
        dim3 dimGrid(_ml->nrncuda_info->num_blocks, 1, 1);

        gpu_init_kernel<<<dimGrid, dimBlock>>>(_nt->nrncuda_defines, *_ml->nrncuda_info);
        err = hipGetLastError();
 
	// printf("Done with pasx init\n");

}

//#define _threadargscomma_ _p, _ppvar, _cljarg _thread, _nt,
#define _threadargscommakern_ _p, _ppvar, _clj,
//#define _threadargsprotocomma_ double* _p, Datum* _ppvar, _cljproto Datum* _thread, _NrnThread* _nt,
#define _threadargsprotocommakern_ double* _p, Datum* _ppvar, int _clj, 
/*
__device__ void _pasx_current(_threadargsprotocommakern_ double* _current){ 
    *_current = 0.;
    i = g * ( v - e );
    *_current += i;
}
*/
#define _pasx_current(_current) \
    *_current = 0.; \
    i = g * ( v - e ); \
    *_current += i; \

__global__ void
gpu_pasx_cur_kernel(nrncuda_defines_t dp, nrncuda_memb_prop_t mp)
{
	double _rhs;
	Datum* _ppvar;
        _cldec

	double lv;
	double li;
	double lg;
	double l_rhs;
	double le;
	double l_g;

        int _pod=(blockIdx.x*GPU_ADVANCE_BLOCKSIZE/GPU_PODSIZE) + (threadIdx.x >> GPU_LOGPODSIZE);
	if (_pod >= mp.num_pods) return;
        _clj  =  threadIdx.x & (GPU_PODSIZE-1);
        int podoffset = GPU_PODSIZE * _pod;
        double* _p = &dp._params[mp.param_start_offset + podoffset*5];
        int node = dp._nodeindices[mp.node_start_offset + podoffset + _clj];

        lv = dp.VEC._v[node];
	lv += 0.001;

	lg = g;
	le = e;
        l_g = lg * ( lv - le );
	lv -= 0.001;
        l_rhs = lg * ( lv - le );
        _g = (l_g - l_rhs)*1000.0; 

	atomicAdd(&dp.VEC._rhs[node], -l_rhs);

/*
        v = dp.VEC._v[node];
	v += 0.001;
	_pasx_current(_threadargscommakern_ &_g);
	v -= 0.001;
	_pasx_current(_threadargscommakern_ &_rhs);
        _g = (_g - _rhs)*1000.0; 

	atomicAdd(&dp.VEC._rhs[node], -_rhs);
*/
}

static void nrn_cur(_NrnThread* _nt, _Memb_list* _ml, int _type){
        hipError_t err;
        int num_blocks;
        int num;

        dim3 dimBlock(GPU_ADVANCE_BLOCKSIZE, 1, 1);
        dim3 dimGrid(_ml->nrncuda_info->num_blocks, 1, 1);

        err = hipMemcpy( _nt->nrncuda_defines.VEC._rhs,  _nt->_actual_rhs,
                          _nt->nrncuda_defines.VEC._size_rhs, hipMemcpyHostToDevice);
        if (err != hipSuccess) return;
        gpu_pasx_cur_kernel<<<dimGrid, dimBlock>>>(_nt->nrncuda_defines, *_ml->nrncuda_info);
        err = hipGetLastError();
        if (err != hipSuccess) return;
        err = hipMemcpy( _nt->_actual_rhs, _nt->nrncuda_defines.VEC._rhs,
                          _nt->nrncuda_defines.VEC._size_rhs, hipMemcpyDeviceToHost);
        if (err != hipSuccess) return;
 
	// printf("Done with pasx cur\n");

}
/*
static void _nrn_current(_threadargsprotocomma_ double* _current){ _clb _current[_clj] = 0.; _cle
{ {
   _clb i = g * ( v - e ) ;
   _cle }
 _clb _current[_clj] += i; _cle
}}

static void nrn_cur(_NrnThread* _nt, _Memb_list* _ml, int _type) {
double* _p; Datum* _ppvar; Datum* _thread; _cldec0
Node *_nd; int* _ni; double _rhs[_cls]; int _iml, _cntml;
#if CACHEVEC
    _ni = _ml->_nodeindices;
#endif
_cntml = _ml->_nodecount;
_thread = _ml->_thread;
for (_iml = 0; _iml < _cntml; ++_iml) {
 _p = _ml->_data[_iml]; _ppvar = _ml->_pdata[_iml];
 _clb v = VEC_V(_ni[_iml*_cls + _clj]); _cle
 _clb v += .001; _cle;
 {_cldec0 _nrn_current(_threadargscomma_ &_g);}
 _clb v -= .001; _cle;
{ {_cldec0 _nrn_current(_threadargscomma_ _rhs);}
 	}
 _clb _g = (_g - _rhs[_clj])/.001; _cle
 _clb VEC_RHS(_ni[_iml*_cls + _clj]) -= _rhs[_clj]; _cle
  printf("Done with nrn_cur in pasx.cu\n");
 
}}
*/

__global__ void
gpu_pasx_jacob_kernel(nrncuda_defines_t dp, nrncuda_memb_prop_t mp)
{
	double _rhs;
	Datum* _ppvar;
        _cldec

        int _pod  = (blockIdx.x*GPU_ADVANCE_BLOCKSIZE/GPU_PODSIZE) + (threadIdx.x >> GPU_LOGPODSIZE);
	if (_pod >= mp.num_pods) return;
        _clj  =  threadIdx.x & (GPU_PODSIZE-1);
        int podoffset = GPU_PODSIZE * _pod;
        double* _p = &dp._params[mp.param_start_offset + podoffset*5];
        int node = dp._nodeindices[mp.node_start_offset + podoffset + _clj];
        atomicAdd(&dp.VEC._d[node], _g);
}

static void nrn_jacob(_NrnThread* _nt, _Memb_list* _ml, int _type){
        hipError_t err;
        int num_blocks;
        int num;

        dim3 dimBlock(GPU_ADVANCE_BLOCKSIZE, 1, 1);
        dim3 dimGrid(_ml->nrncuda_info->num_blocks, 1, 1);

        err = hipMemcpy( _nt->nrncuda_defines.VEC._d,  _nt->_actual_d,
                          _nt->nrncuda_defines.VEC._size_d, hipMemcpyHostToDevice);
        if (err != hipSuccess) return;
        gpu_pasx_jacob_kernel<<<dimGrid, dimBlock>>>(_nt->nrncuda_defines, *_ml->nrncuda_info);
        err = hipGetLastError();
        if (err != hipSuccess) return;
        err = hipMemcpy( _nt->_actual_d, _nt->nrncuda_defines.VEC._d,
                          _nt->nrncuda_defines.VEC._size_d, hipMemcpyDeviceToHost);
        if (err != hipSuccess) return;

        // printf("Done with pasx jacob\n");

}

/*
static void nrn_jacob(_NrnThread* _nt, _Memb_list* _ml, int _type) {
double* _p; Datum* _ppvar; Datum* _thread; _cldec0
Node *_nd; int* _ni; int _iml, _cntml;
#if CACHEVEC
    _ni = _ml->_nodeindices;
#endif
_cntml = _ml->_nodecount;
_thread = _ml->_thread;
for (_iml = 0; _iml < _cntml; ++_iml) {
 _p = _ml->_data[_iml];
#if CACHEVEC
  if (use_cachevec) {
	_clb VEC_D(_ni[_iml*_cls + _clj]) += _g; _cle
  }else
#endif
  {
     _clb _nd = _ml->_nodelist[_iml*_cls + _clj];
	NODED(_nd) += _g; _cle
  }
 
  printf("Done with nrn_jacob in pasx.cu\n");
}}
*/

__global__ void
gpu_pasx_state_kernel(nrncuda_defines_t dp, nrncuda_memb_prop_t mp)
{
	double _rhs;
	Datum* _ppvar;
        _cldec

        int _pod  = (blockIdx.x*GPU_ADVANCE_BLOCKSIZE/GPU_PODSIZE) + (threadIdx.x >> GPU_LOGPODSIZE);
	if (_pod >= mp.num_pods) return;
        _clj  =  threadIdx.x & (GPU_PODSIZE-1);
        int podoffset = GPU_PODSIZE * _pod;
        double* _p = &dp._params[mp.param_start_offset + podoffset*5];
        int node = dp._nodeindices[mp.node_start_offset + podoffset + _clj];
}

static void nrn_state(_NrnThread* _nt, _Memb_list* _ml, int _type){
        hipError_t err;
        int num_blocks;
        int num;

        dim3 dimBlock(GPU_ADVANCE_BLOCKSIZE, 1, 1);
        dim3 dimGrid(_ml->nrncuda_info->num_blocks, 1, 1);

        gpu_pasx_state_kernel<<<dimGrid, dimBlock>>>(_nt->nrncuda_defines, *_ml->nrncuda_info);
        err = hipGetLastError();

        // printf("Done with pasx states\n");

}

/*
static void nrn_state(_NrnThread* _nt, _Memb_list* _ml, int _type) {

  printf("Done with nrn_state in pasx.cu\n");
}
*/

static void terminal(){}

static void _initlists(){
 double _x; double* _p = &_x; _cldec0
 int _i; static int _first = 1;
  if (!_first) return;
_first = 0;
}
}
